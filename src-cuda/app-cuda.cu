#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdint>
#include <iostream>
#include <memory>
#include <stdexcept> // runtime_error
#include <string>
#include <utility>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>

// https://github.com/nothings/stb/blob/master/stb_image.h#L12
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
// https://github.com/nothings/stb/blob/master/stb_image_write.h#L7
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

struct Color {
    uint8_t r, g, b;
};

struct square {
    __device__ double operator()(const uint8_t& x) const {
        return double(x) * double(x);
    }
};

struct RgbSoa {
    thrust::device_vector<uint8_t> r, g, b;
    RgbSoa(const thrust::host_vector<uint8_t>& r, const thrust::host_vector<uint8_t>& g, const thrust::host_vector<uint8_t>& b) : r(r), g(g), b(b) {
    }
};

constexpr unsigned MAX_DEPTH = 8;
constexpr unsigned DETAIL_THRESHOLD = 13;

uint8_t* read_file(std::string filename, int* w, int* h, int* n) {
    uint8_t* data = stbi_load(filename.c_str(), w, h, n, 3);
    if (!data) {
        throw std::runtime_error("Could not open file 'images/funny.jpg'");
    }
    return data;
}

class Quadtree {
  public:
    std::unique_ptr<Quadtree> nw;
    std::unique_ptr<Quadtree> ne;
    std::unique_ptr<Quadtree> se;
    std::unique_ptr<Quadtree> sw;

    unsigned depth;
    unsigned x, y;
    unsigned h, w;

    Color color;

    // Constructs the root of a quadtree
    Quadtree(unsigned h, unsigned w)
        : depth(0), x(0), y(0), h(h), w(h) {
#ifndef NDEBUG
        std::cout << "Create quadtree root\n";
#endif
    }

    // Constructs an internal node of a quadtree. Todo: make private
    Quadtree(unsigned depth, unsigned x, unsigned y, unsigned h, unsigned w)
        : depth(depth), x(x), y(y), h(h), w(h) {
#ifndef NDEBUG
        std::cout << "Create quadtree node at depth " << depth << '\n';
#endif
    }

    void build_quadtree(const RgbSoa& image, unsigned left, unsigned right);

    unsigned n_pixels() const {
        return h * w;
    }
};

void Quadtree::build_quadtree(const RgbSoa& image, unsigned left, unsigned right) {
    auto should_split_start = std::chrono::steady_clock::now();

    auto r_sum = thrust::reduce(image.r.begin() + left, image.r.begin() + right, (double)0);
    auto g_sum = thrust::reduce(image.g.begin() + left, image.g.begin() + right, (double)0);
    auto b_sum = thrust::reduce(image.b.begin() + left, image.b.begin() + right, (double)0);

    auto r_sq_sum = thrust::transform_reduce(image.r.begin() + left, image.r.begin() + right, square(), (double)0, thrust::plus<double>());
    auto g_sq_sum = thrust::transform_reduce(image.g.begin() + left, image.g.begin() + right, square(), (double)0, thrust::plus<double>());
    auto b_sq_sum = thrust::transform_reduce(image.b.begin() + left, image.b.begin() + right, square(), (double)0, thrust::plus<double>());

    double r_mean = r_sum / image.r.size();
    double g_mean = g_sum / image.g.size();
    double b_mean = b_sum / image.b.size();

    double r_sq_mean = r_sq_sum / image.r.size();
    double g_sq_mean = g_sq_sum / image.g.size();
    double b_sq_mean = b_sq_sum / image.b.size();

    double r_stdev = std::sqrt(r_sq_mean - std::pow(r_mean, 2));
    double g_stdev = std::sqrt(g_sq_mean - std::pow(g_mean, 2));
    double b_stdev = std::sqrt(b_sq_mean - std::pow(b_mean, 2));

    auto should_split_end = std::chrono::steady_clock::now();
#ifndef NDEBUG
    std::cout << "Determine if split time: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(should_split_end - should_split_start).count()
              << " ms\n";
#endif

    if (depth < MAX_DEPTH &&
        r_stdev > DETAIL_THRESHOLD &&
        g_stdev > DETAIL_THRESHOLD &&
        b_stdev > DETAIL_THRESHOLD) {

        nw = std::make_unique<Quadtree>(depth + 1, x, y, h / 2, w / 2);
        ne = std::make_unique<Quadtree>(depth + 1, x, y + w / 2, h / 2, w / 2);
        sw = std::make_unique<Quadtree>(depth + 1, x + h / 2, y, h / 2, w / 2);
        se = std::make_unique<Quadtree>(depth + 1, x + h / 2, y + w / 2, h / 2, w / 2);

        unsigned dim = (right - left) / 4;
        nw->build_quadtree(image, left + 0 * dim, left + 1 * dim);
        ne->build_quadtree(image, left + 1 * dim, left + 2 * dim);
        sw->build_quadtree(image, left + 2 * dim, left + 3 * dim);
        se->build_quadtree(image, left + 3 * dim, left + 4 * dim);
    } else {
        color = {(uint8_t)r_mean, (uint8_t)g_mean, (uint8_t)b_mean};
    }
}

void flatten_data(uint8_t* data,
                  thrust::host_vector<uint8_t>& r,
                  thrust::host_vector<uint8_t>& g,
                  thrust::host_vector<uint8_t>& b,
                  int top,
                  int left,
                  int w,
                  int h,
                  int& i,
                  int W) {
    if (w == 1) {
        r[i] = data[(top * W + left) * 3 + 0];
        g[i] = data[(top * W + left) * 3 + 1];
        b[i] = data[(top * W + left) * 3 + 2];
        i++;
    } else {
        flatten_data(data, r, g, b, top, left, w / 2, h / 2, i, W);
        flatten_data(data, r, g, b, top, left + w / 2, w / 2, h / 2, i, W);
        flatten_data(data, r, g, b, top + h / 2, left, w / 2, h / 2, i, W);
        flatten_data(data, r, g, b, top + h / 2, left + w / 2, w / 2, h / 2, i, W);
    }
}

void unflatten_data(uint8_t* data,
                    thrust::host_vector<uint8_t>& r,
                    thrust::host_vector<uint8_t>& g,
                    thrust::host_vector<uint8_t>& b,
                    int top,
                    int left,
                    int w,
                    int h,
                    int& i,
                    int W) {
    if (w == 1) {
        data[(top * W + left) * 3 + 0] = r[i];
        data[(top * W + left) * 3 + 1] = g[i];
        data[(top * W + left) * 3 + 2] = b[i];
        i++;
    } else {
        unflatten_data(data, r, g, b, top, left, w / 2, h / 2, i, W);
        unflatten_data(data, r, g, b, top, left + w / 2, w / 2, h / 2, i, W);
        unflatten_data(data, r, g, b, top + h / 2, left, w / 2, h / 2, i, W);
        unflatten_data(data, r, g, b, top + h / 2, left + w / 2, w / 2, h / 2, i, W);
    }
}

int main() {
    int w, h, n;
    uint8_t* data = read_file("images/funny.jpg", &w, &h, &n);
    int n_pixels = w * h;
    thrust::host_vector<uint8_t> r(n_pixels);
    thrust::host_vector<uint8_t> g(n_pixels);
    thrust::host_vector<uint8_t> b(n_pixels);
    // fill red, green, blue color vectors
    int i = 0;
    flatten_data(data, r, g, b, 0, 0, w, h, i, w);
    // for (int i = 0; i < n_pixels / 3; i++) {
    //     r[i] = data[i * 3 + 0];
    //     g[i] = data[i * 3 + 1];
    //     b[i] = data[i * 3 + 2];
    // }
    RgbSoa image(r, g, b);

    std::cout << "Start quadtree build\n";

    auto start = std::chrono::steady_clock::now();

    Quadtree root(h, w);
    root.build_quadtree(image, 0, n_pixels);
    // TODO: set in r, g, b the new color

    auto end = std::chrono::steady_clock::now();
    std::cout << "Elapsed time in milliseconds: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count()
              << " ms\n";

    i = 0;
    unflatten_data(data, r, g, b, 0, 0, w, h, i, w);
    stbi_write_jpg("result.jpg", w, h, 3, data, 100);
    stbi_image_free(data);

    return 0;
}
