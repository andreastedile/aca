#include "colorization.h"
#include "construction_device.cuh"
#include "construction_host.h"
#include "cuda_check.h"
#include "io.h"
#include "node.h"
#include "padding.h"
#include "qtmath.h"
#include "rgbaos.h"

#include <argparse/argparse.hpp>
#include <chrono>
#include <hip/hip_runtime.h>
#include <fstream>
#include <omp.h>
#include <spdlog/spdlog.h>

using Clock = std::chrono::steady_clock;
using ms = std::chrono::milliseconds;

int main(int argc, char* argv[]) {
    spdlog::set_level(spdlog::level::debug);

    argparse::ArgumentParser app("qt-it-gpu");

    app.add_argument("--input")
        .required()
        .help("specify the input file");
    app.add_argument("--detail-threshold")
        .scan<'g', float>()
        .default_value(13.0f)
        .help("specify the detail threshold");
    app.add_argument("-b", "--blocks")
        .scan<'d', int>()
        .default_value(1024)
        .help("specify the number of blocks in the kernel launch");
    app.add_argument("-t", "--threads")
        .scan<'d', int>()
        .default_value(256)
        .help("specify the number of thread per blocks in the kernel launch");
    app.add_argument("--no-output-file")
        .default_value(false)
        .implicit_value(true)
        .help("suppress the production of the resulting image");
    app.add_argument("--csv")
        .default_value(false)
        .implicit_value(true)
        .help("writes timing information to csv file");

    app.parse_args(argc, argv);

    auto input = app.get("--input");
    auto detail_threshold = app.get<float>("--detail-threshold");
    auto n_blocks = app.get<int>("--blocks");
    auto n_threads = app.get<int>("--threads");
    auto no_output_file = app.get<bool>("--no-output-file");
    auto csv = app.get<bool>("--csv");

    spdlog::info("Read {}", input);
    int n_rows, n_cols, n = 0;
    unsigned char* pixels = read_image(input, n_rows, n_cols);

    int n_pixels = n_rows * n_cols;

    spdlog::info("Flatten to RGB AoS");
    auto flatten_start = Clock::now();
    const auto h_aos = flatten_to_rgb_aos(pixels, n_rows, n_cols);
    auto flatten_end = Clock::now();

    int tree_height = log4(n_pixels);
    int n_nodes = (pow_base4(tree_height + 1) - 1) / 3;
    int n_leaves_per_thread = n_pixels / (n_blocks * n_threads);

    spdlog::info("Allocate RGB AoS and quadtree array on device");
    auto allocate_device_start = Clock::now();
    RGB<unsigned char>* d_aos_aos;
    Node* d_quadtree_array;
    CHECK(hipMalloc(&d_aos_aos, n_pixels * sizeof(RGB<unsigned char>)));
    CHECK(hipMalloc(&d_quadtree_array, n_nodes * sizeof(Node)));
    RGBAoS d_aos(d_aos_aos, n_pixels);
    auto allocate_device_end = Clock::now();

    spdlog::info("Copy RGB AoS on the device");
    auto copy_to_device_start = Clock::now();
    CHECK(hipMemcpy(d_aos_aos, h_aos.aos, n_pixels * sizeof(RGB<unsigned char>), hipMemcpyHostToDevice));
    auto copy_to_device_end = Clock::now();

    delete[] h_aos.aos;

    spdlog::info("Construct quadtree array on device (launch: {}/{}, n leaves per thread: {})", n_blocks, n_threads, n_leaves_per_thread);
    auto construct_device_start = Clock::now();
    construct_quadtree_array_device<<<n_blocks, n_threads>>>(d_quadtree_array, tree_height, d_aos, detail_threshold, n_leaves_per_thread);
    CHECK(hipPeekAtLastError());
    CHECK(hipDeviceSynchronize());
    auto construct_device_end = Clock::now();

    spdlog::info("Allocate quadtree array on host (size in bytes: {})", (size_t)n_nodes * (sizeof(Node)));
    auto allocate_on_host_start = Clock::now();
    Node* h_quadtree_array = new Node[n_nodes];
    auto allocate_on_host_end = Clock::now();

    spdlog::info("Copying quadtree array back to host");
    auto copy_to_host_start = Clock::now();
    CHECK(hipMemcpy(h_quadtree_array, d_quadtree_array, n_nodes * sizeof(Node), hipMemcpyDeviceToHost));
    auto copy_to_host_end = Clock::now();

    CHECK(hipFree(d_aos_aos));
    CHECK(hipFree(d_quadtree_array));

    int from_level = log4(n_blocks) - 1;
    spdlog::info("Construct quadtree array on host from level {}", from_level);
    omp_set_num_threads(8);
    auto construct_on_host_start = Clock::now();
    construct_quadtree_array_host(h_quadtree_array, from_level, detail_threshold);
    auto construct_on_host_end = Clock::now();

    if (csv) {
        std::ofstream file("timings.csv");
        file << "flatten_ms, allocate_device_ms, copy_to_device_ms, construct_on_device_ms, allocate_on_host_ms, construct_on_host_ms\n";
        file << std::chrono::duration_cast<ms>(flatten_end - flatten_start).count() << ", ";
        file << std::chrono::duration_cast<ms>(allocate_device_end - allocate_device_start).count() << ", ";
        file << std::chrono::duration_cast<ms>(copy_to_device_end - copy_to_host_start).count() << ", ";
        file << std::chrono::duration_cast<ms>(construct_device_end - construct_device_start).count() << ", ";
        file << std::chrono::duration_cast<ms>(allocate_on_host_end - allocate_on_host_start).count() << ", ";
        file << std::chrono::duration_cast<ms>(construct_on_host_end - construct_on_host_start).count() << '\n';
    }

    if (!no_output_file) {
        for (int i = 0; i <= tree_height; i++) {
            spdlog::info("Colorize level {}", i);
            colorize(pixels, n_rows, n_cols, h_quadtree_array, i);

            spdlog::info("Write image");
            auto output = std::string("level") + std::to_string(i) + std::string(".jpg");
            write_image(pixels, output, n_rows, n_cols);
        }
    }

    delete[] h_quadtree_array;
    delete[] pixels;

    spdlog::info("All done");

    return 0;
}
