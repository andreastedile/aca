#include "hip/hip_runtime.h"
#include "combination.h"
#include "qtmath.h"

__device__ __host__ bool should_merge(float detail_threshold, const RGB<float>& std) {
    return std.r <= detail_threshold &&
           std.g <= detail_threshold &&
           std.b <= detail_threshold;
}

// source
// https://stats.stackexchange.com/questions/25848/how-to-sum-a-standard-deviation/442050#442050
__device__ __host__ RGB<float> combine_means(const Node& nw, const Node& ne, const Node& se, const Node& sw) {
    // int n_pixels = 4 * n_pixels_subquadrant;
    RGB<float> nw_mean = nw.mean,
               ne_mean = ne.mean,
               se_mean = se.mean,
               sw_mean = sw.mean;
    return {
        (nw_mean.r + ne_mean.r + se_mean.r + sw_mean.r) / 4,
        (nw_mean.g + ne_mean.g + se_mean.g + sw_mean.g) / 4,
        (nw_mean.b + ne_mean.b + se_mean.b + sw_mean.b) / 4,
    };
    // return {
    //     (nw_mean.r * n_pixels_subquadrant + ne_mean.r * n_pixels_subquadrant + se_mean.r * n_pixels_subquadrant + sw_mean.r * n_pixels_subquadrant) / n_pixels,
    //     (nw_mean.g * n_pixels_subquadrant + ne_mean.g * n_pixels_subquadrant + se_mean.g * n_pixels_subquadrant + sw_mean.g * n_pixels_subquadrant) / n_pixels,
    //     (nw_mean.b * n_pixels_subquadrant + ne_mean.b * n_pixels_subquadrant + se_mean.b * n_pixels_subquadrant + sw_mean.b * n_pixels_subquadrant) / n_pixels,
    // };
}

__device__ __host__ RGB<float> combine_stds(const Node& nw, const Node& ne, const Node& se, const Node& sw, const RGB<float>& mean) {
    // int n_pixels = 4 * n_pixels_subquadrant;
    RGB<float> nw_mean = nw.mean,
               ne_mean = ne.mean,
               se_mean = se.mean,
               sw_mean = sw.mean;
    RGB<float> nw_std = nw.std,
               ne_std = ne.std,
               se_std = se.std,
               sw_std = sw.std;

    auto combine = [](float nw_std, float ne_std, float se_std, float sw_std,
                      float nw_mean, float ne_mean, float se_mean, float sw_mean,
                      float mean) {
        return sqrtf(
                   square(nw_std) + square(mean - nw_mean) +
                   square(ne_std) + square(mean - ne_mean) +
                   square(se_std) + square(mean - se_mean) +
                   square(sw_std) + square(mean - sw_mean)) /
               2;
    };

    return {
        combine(nw_std.r, ne_std.r, se_std.r, sw_std.r, nw_mean.r, ne_mean.r, se_mean.r, sw_mean.r, mean.r),
        combine(nw_std.g, ne_std.g, se_std.g, sw_std.g, nw_mean.g, ne_mean.g, se_mean.g, sw_mean.g, mean.g),
        combine(nw_std.b, ne_std.b, se_std.b, sw_std.b, nw_mean.b, ne_mean.b, se_mean.b, sw_mean.b, mean.b),
    };
}

__device__ __host__ Node make_internal_node(Node& nw, Node& ne, Node& se, Node& sw, float detail_threshold) {
    auto mean = combine_means(nw, ne, se, sw);
    auto std = combine_stds(nw, ne, se, sw, mean);
    if (should_merge(detail_threshold, std)) {
        return {mean, std, Node::Type::LEAF};
    } else {
        return {mean, std, Node::Type::FORK};
    }
}
