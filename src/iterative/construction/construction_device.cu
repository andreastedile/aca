#include "hip/hip_runtime.h"
#include "combination.h"
#include "construction_device.cuh"
#include "qtmath.h"

__device__ void init_quadtree_array_leaves_device(RGBAoS aos, Node* quadtree_nodes, int tree_height, int n_leaves_per_thread) {
    int n_higher_nodes = (pow_base4(tree_height) - 1) / 3;
    int block_offset = blockIdx.x * blockDim.x;

    auto read_ptr = aos.aos + n_leaves_per_thread * (block_offset + threadIdx.x);

    Node* current_thread_leaves = quadtree_nodes +
                                  n_higher_nodes +
                                  n_leaves_per_thread * (block_offset + threadIdx.x);

#pragma unroll
    for (int i = 0; i < n_leaves_per_thread; i++) {
        current_thread_leaves[i] = Node(
            {float(read_ptr[i].r), float(read_ptr[i].g), float(read_ptr[i].b)}, // mean
            {0.0, 0.0, 0.0},                                                    // std
            Node::Type::LEAF);
    }
}

__global__ void construct_quadtree_array_device(Node* g_nodes, int tree_height, RGBAoS aos, float detail_threshold, int n_leaves_per_thread) {
    init_quadtree_array_leaves_device(aos, g_nodes, tree_height, n_leaves_per_thread);
    __syncthreads();

    // Number of pixels contained in the subquadrant represented by a node at the level immediately below.
    // Below this level are the leaves, each representing 1 pixel.
    int n_nodes_at_higher_levels = (pow_base4(tree_height - 1) - 1) / 3;
    int current_level_n_nodes = pow_base4(tree_height - 1);
    Node* lower_level_nodes = g_nodes + n_nodes_at_higher_levels + current_level_n_nodes;
    Node* current_level_nodes = g_nodes + n_nodes_at_higher_levels;

#pragma unroll
    for (int n_nodes_to_produce = div4(n_leaves_per_thread);
         n_nodes_to_produce >= 1;
         n_nodes_to_produce = div4(n_nodes_to_produce)) {
        int thread_nodes_idx = n_nodes_to_produce * (blockIdx.x * blockDim.x + threadIdx.x);

#pragma unroll
        for (int i = 0; i < n_nodes_to_produce; i++) {
            int node_idx = thread_nodes_idx + i;
            int children_idx = times4(node_idx);
            current_level_nodes[node_idx] = make_internal_node(
                lower_level_nodes[children_idx + 0],
                lower_level_nodes[children_idx + 1],
                lower_level_nodes[children_idx + 2],
                lower_level_nodes[children_idx + 3],
                detail_threshold);
        }
        current_level_n_nodes = div4(current_level_n_nodes);
        lower_level_nodes = current_level_nodes;
        current_level_nodes = current_level_nodes - current_level_n_nodes;
    }

    __syncthreads();

    // Above we handle the cases where each threads produces at least one node.

    // We enter the following for when there are more threads than nodes to produce.

    for (
        // At the current level, in the current block, each threads reduce four nodes at the level immediately below
        int n_working_threads_per_block = div4(blockDim.x),
            // The current level in the tree that we are constructing
        curr_level = tree_height - 1;
        // We need a thread to reduce four nodes.
        n_working_threads_per_block >= 1;
        // Equivalently, we can terminate the construction when the number of nodes at the current level amounts to the number of blocks.
        // curr_level >= tree_height - log4(blockDim.x),
        n_working_threads_per_block = div4(n_working_threads_per_block),
            curr_level--) {
        if (threadIdx.x < n_working_threads_per_block) {
            int node_idx = blockIdx.x * n_working_threads_per_block + threadIdx.x;
            int children_idx = times4(node_idx);
            current_level_nodes[node_idx] = make_internal_node(
                lower_level_nodes[children_idx + 0],
                lower_level_nodes[children_idx + 1],
                lower_level_nodes[children_idx + 2],
                lower_level_nodes[children_idx + 3],
                detail_threshold);
        }

        __syncthreads();

        current_level_n_nodes = div4(current_level_n_nodes);
        lower_level_nodes = current_level_nodes;
        current_level_nodes = current_level_nodes - current_level_n_nodes;
    }
}
