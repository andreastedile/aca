#include "hip/hip_runtime.h"
#include "qtmath.h"

#include <cmath>

// compute 2*n
CUDA_HOSTDEV int times2(int n) {
    return n << 1;
}

// compute 2^n
CUDA_HOSTDEV int pow_base2(int n) {
    return 1 << n;
}

// compute 4^n = 2^(2n)
CUDA_HOSTDEV int pow_base4(int n) {
    return pow_base2(times2(n));
}

CUDA_HOSTDEV int div2(int n) {
    return n >> 1;
}

CUDA_HOSTDEV int div4(int n) {
    return n >> 2;
}

CUDA_HOSTDEV int times4(int n) {
    return n << 2;
}

CUDA_HOSTDEV int square(int n) {
    return n * n;
}

CUDA_HOSTDEV int log4(int n) {
    return div2(int(std::log2(n)));
}

CUDA_HOSTDEV bool is_pow2(int n) {
    return n == std::pow(2, int(std::log2(n)));
}

CUDA_HOSTDEV bool is_pow4(int n) {
    return n == std::pow(4, log4(n));
}
